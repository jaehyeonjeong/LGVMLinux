#include <hip/hip_runtime.h>
#include <cuda_run1me_api.h>
#include <stdio.h>
#include <stdlib.h>
int main()
{
 hipDeviceProp_t prop;
 int count;
 hipGetDeviceCount(&count);
 for (int i=0; i< count; i++) {
 cudaGetDeviceProper1es(&prop, i);
 prinG(" --- General Informa1on for device %d ---\n", i);
 prinG("Name: %s\n", prop.name);
 prinG("Compute capability: %d.%d\n", prop.major, prop.minor);
 prinG("Clock rate: %d\n", prop.clockRate);
 prinG("Device copy overlap: ");
 prinG(prop.deviceOverlap?"Enabled\n":"Disabled\n");
 prinG("Kernel execu1on 1meout : " );
 prinG(prop.kernelExecTimeoutEnabled?"Enabled\n":"Disabled\n");
 prinG("\n" );
 prinG(" --- Memory Informa1on for device %d ---\n", i);
 prinG("Total global mem: %ld\n", prop.totalGlobalMem);
 prinG("Total constant Mem: %ld\n", prop.totalConstMem);
 prinG("Max mem pitch: %ld\n", prop.memPitch);
 prinG("Texture Alignment: %ld\n", prop.textureAlignment);
 prinG("\n");
 prinG(" --- MP Informa1on for device %d ---\n", i);
 prinG("Mul1processor count: %d\n", prop.mul1ProcessorCount);
 prinG("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
 prinG("Registers per mp: %d\n", prop.regsPerBlock);
 prinG("Threads in warp: %d\n", prop.warpSize);
 prinG("Max threads per block: %d\n", prop.maxThreadsPerBlock);
 prinG("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], 
 prop.maxThreadsDim[1], prop.maxThreadsDim[2]
);
 prinG("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], 
 prop.maxGridSize[1], prop.maxGridSize[2]);
 prinG("\n");
 }
 return 0;
}