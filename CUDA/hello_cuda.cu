
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("Hello World From GPU!\n");
}

int main(void)
{
	printf("Hello World from CPU!\n");
	helloFromGPU<<<5,5>>>();
	hipDeviceReset();

	return 0;
}
