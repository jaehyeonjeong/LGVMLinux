#include <hip/hip_runtime.h>
#include <cuda_run1me_api.h>
#include <stdio.h>
#include <stdlib.h>
// This func1on wraps the CUDA Driver API into a template func1on
template <class T>
inline void getCudaADribute(T *aDribute, CUdevice_aDribute device_aDribute, 
 int device)
{
 hipError_t error = cuDeviceGetADribute(aDribute, device_aDribute, device);
 if(hipSuccess != error) {
 fprinP(stderr, "cuSafeCallNoSync( ) Driver API error = %04d from file <%s>, line %i.\n", err
or, __FILE__, __LINE__);
 exit(-1);
 }
}
int main() 
{
 prinP("CUDA Version %i\n", CUDA_VERSION);
 prinP("CUDA Version %s\n", VERSION_STATUS);
 prinP("\nDriver\n");
 hipInit(0);
 int driverVersion, run1meVersion;
 hipDriverGetVersion(&driverVersion);
 cudaRun1meGetVersion(&run1meVersion);
 prinP(" CUDA Version : %i\n", driverVersion);
 prinP(" Run1me version : %i\n", run1meVersion);
 int deviceCount = 0;
 hipError_t error_id = hipGetDeviceCount(&deviceCount);
 prinP(" Number of device : %i\n", deviceCount);
 int value;
 hipDeviceProp_t prop;
for(int id = 0; id < deviceCount; id++) {
 cudaGetDeviceProper1es(&prop, id);
 prinP(" Device Name : %s\n", prop.name);
 prinP(" Total global mem : %ld\n", prop.totalGlobalMem);
 prinP(" Total Constant Mem : %ld\n", prop.totalConstMem);
 prinP("\n ADribute(%i)\n", id+1);
 getCudaADribute<int>(&value, hipDeviceAttributeMaxThreadsPerBlock, id);
 prinP(" Max Threads per Block : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeMaxBlockDimX, id);
 prinP(" Max Block DIM(x) : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeMaxBlockDimY, id);
 prinP(" Max Block DIM(y) : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeMaxBlockDimZ, id);
 prinP(" Max Block DIM(z) : %i\n", value);
getCudaADribute<int>(&value, hipDeviceAttributeTotalConstantMemory, id);
 prinP(" Total Constant Memory : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeWarpSize, id);
 prinP(" Warp size : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeMemoryClockRate, id);
 prinP(" Clock Rate : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeMemoryBusWidth, id);
 prinP(" Memory Bus Width : %i\n", value);
 getCudaADribute<int>(&value, hipDeviceAttributeL2CacheSize, id);
 prinP(" L2 Cache Size : %i\n", value);
 }
 return EXIT_SUCCESS;
}